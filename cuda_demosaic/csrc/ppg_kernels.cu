#include "hip/hip_runtime.h"
/*
 * PPG Demosaic CUDA Kernels
 * Converted from OpenCL kernels for better PyTorch integration
 */

#include <hip/hip_runtime.h>
#include <>
#include <torch/types.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>
#include <cstdint>

// FC macro for Bayer pattern (from darktable)
#define FC(row, col, filters) ((filters >> ((((row) << 1 & 14) + ((col) & 1)) << 1)) & 3)

__constant__ int glim[5] = { 0, 1, 2, 1, 0 };

#define SWAP(a, b)                \
  {                               \
    const float tmp = (b);        \
    (b) = (a);                    \
    (a) = tmp;                    \
  }

/**
 * Pre-median filtering kernel - exact translation from darktable OpenCL
 */
__global__ void pre_median_kernel(
    float* input,
    float* output, 
    int width,
    int height,
    uint32_t filters,
    float threshold
) {
    extern __shared__ float median_buffer[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xlsz = blockDim.x;
    const int ylsz = blockDim.y;
    const int xlid = threadIdx.x;
    const int ylid = threadIdx.y;
    const int xgid = blockIdx.x;
    const int ygid = blockIdx.y;

    // individual control variable in this work group and the work group size
    const int l = ylid * xlsz + xlid;
    const int lsz = xlsz * ylsz;

    // stride and maximum capacity of local buffer
    // cells of 1*float per pixel with a surrounding border of 2 cells
    const int stride = xlsz + 2*2;
    const int maxbuf = stride * (ylsz + 2*2);

    // coordinates of top left pixel of buffer
    // this is 2 pixel left and above of the work group origin
    const int xul = xgid * xlsz - 2;
    const int yul = ygid * ylsz - 2;

    // populate local memory buffer
    for(int n = 0; n <= maxbuf/lsz; n++)
    {
        const int bufidx = n * lsz + l;
        if(bufidx >= maxbuf) continue;
        const int xx = xul + bufidx % stride;
        const int yy = yul + bufidx / stride;
        median_buffer[bufidx] = (xx >= 0 && yy >= 0 && xx < width && yy < height) ? input[yy * width + xx] : 0.0f;
    }

    // center buffer around current x,y-Pixel
    float* centered_median = median_buffer + (ylid + 2) * stride + xlid + 2;

    __syncthreads();

    if(x >= width || y >= height) return;

    const int *lim = glim;

    const int c = FC(y, x, filters);

    float med[9];

    int cnt = 0;

    for(int k = 0, i = 0; i < 5; i++)
    {
        for(int j = -lim[i]; j <= lim[i]; j += 2)
        {
            if(fabsf(centered_median[stride * (i - 2) + j] - centered_median[0]) < threshold)
            {
                med[k++] = centered_median[stride * (i - 2) + j];
                cnt++;
            }
            else
                med[k++] = 64.0f + centered_median[stride * (i - 2) + j];
        }
    }

    for(int i = 0; i < 8; i++)
        for(int ii = i + 1; ii < 9; ii++)
            if(med[i] > med[ii]) SWAP(med[i], med[ii]);

    float color = (c & 1) ? (cnt == 1 ? med[4] - 64.0f : med[(cnt - 1) / 2]) : centered_median[0];

    output[y * width + x] = fmaxf(color, 0.0f);
}

#undef SWAP

/**
 * fill greens pass of pattern pixel grouping - exact translation from darktable OpenCL
 * in (float) or (float4).x -> out (float4)
 */
__global__ void ppg_demosaic_green_kernel(
    float* input,
    float4* output,
    int width,
    int height,
    uint32_t filters
) {
    extern __shared__ float green_buffer[];
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xlsz = blockDim.x;
    const int ylsz = blockDim.y;
    const int xlid = threadIdx.x;
    const int ylid = threadIdx.y;
    const int xgid = blockIdx.x;
    const int ygid = blockIdx.y;

    // individual control variable in this work group and the work group size
    const int l = ylid * xlsz + xlid;
    const int lsz = xlsz * ylsz;

    // stride and maximum capacity of local buffer
    // cells of 1*float per pixel with a surrounding border of 3 cells
    const int stride = xlsz + 2*3;
    const int maxbuf = stride * (ylsz + 2*3);

    // coordinates of top left pixel of buffer
    // this is 3 pixel left and above of the work group origin
    const int xul = xgid * xlsz - 3;
    const int yul = ygid * ylsz - 3;

    // populate local memory buffer
    for(int n = 0; n <= maxbuf/lsz; n++)
    {
        const int bufidx = n * lsz + l;
        if(bufidx >= maxbuf) continue;
        const int xx = xul + bufidx % stride;
        const int yy = yul + bufidx / stride;
        green_buffer[bufidx] = (xx >= 0 && yy >= 0 && xx < width && yy < height) ? input[yy * width + xx] : 0.0f;
    }

    // center buffer around current x,y-Pixel
    float* centered_green = green_buffer + (ylid + 3) * stride + xlid + 3;

    __syncthreads();

    // make sure we dont write the outermost 3 pixels
    if(x >= width - 3 || x < 3 || y >= height - 3 || y < 3) return;
    
    // process all non-green pixels
    const int row = y;
    const int col = x;
    const int c = FC(row, col, filters);
    float4 color = make_float4(0.0f, 0.0f, 0.0f, 1.0f); // output color

    const float pc = centered_green[0];

    if     (c == 0) color.x = pc; // red
    else if(c == 1) color.y = pc; // green1
    else if(c == 2) color.z = pc; // blue
    else            color.y = pc; // green2

    // fill green layer for red and blue pixels:
    if(c == 0 || c == 2)
    {
        // look up horizontal and vertical neighbours, sharpened weight:
        const float pym  = centered_green[-1 * stride];
        const float pym2 = centered_green[-2 * stride];
        const float pym3 = centered_green[-3 * stride];
        const float pyM  = centered_green[ 1 * stride];
        const float pyM2 = centered_green[ 2 * stride];
        const float pyM3 = centered_green[ 3 * stride];
        const float pxm  = centered_green[-1];
        const float pxm2 = centered_green[-2];
        const float pxm3 = centered_green[-3];
        const float pxM  = centered_green[ 1];
        const float pxM2 = centered_green[ 2];
        const float pxM3 = centered_green[ 3];
        const float guessx = (pxm + pc + pxM) * 2.0f - pxM2 - pxm2;
        const float diffx  = (fabsf(pxm2 - pc) +
                              fabsf(pxM2 - pc) +
                              fabsf(pxm  - pxM)) * 3.0f +
                             (fabsf(pxM3 - pxM) + fabsf(pxm3 - pxm)) * 2.0f;
        const float guessy = (pym + pc + pyM) * 2.0f - pyM2 - pym2;
        const float diffy  = (fabsf(pym2 - pc) +
                              fabsf(pyM2 - pc) +
                              fabsf(pym  - pyM)) * 3.0f +
                             (fabsf(pyM3 - pyM) + fabsf(pym3 - pym)) * 2.0f;
        if(diffx > diffy)
        {
            // use guessy
            const float m = fminf(pym, pyM);
            const float M = fmaxf(pym, pyM);
            color.y = fmaxf(fminf(guessy*0.25f, M), m);
        }
        else
        {
            const float m = fminf(pxm, pxM);
            const float M = fmaxf(pxm, pxM);
            color.y = fmaxf(fminf(guessx*0.25f, M), m);
        }
    }
    output[y * width + x] = make_float4(fmaxf(color.x, 0.0f), fmaxf(color.y, 0.0f), fmaxf(color.z, 0.0f), 1.0f);
}

/**
 * fills the reds and blues in the gaps (done after ppg_demosaic_green) - exact translation from darktable OpenCL
 * in (float4) -> out (float4)
 */
__global__ void ppg_demosaic_redblue_kernel(
    float4* input,
    float4* output,
    int width,
    int height,
    uint32_t filters
) {
    extern __shared__ float4 redblue_buffer[];
    // image in contains full green and sparse r b
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int xlsz = blockDim.x;
    const int ylsz = blockDim.y;
    const int xlid = threadIdx.x;
    const int ylid = threadIdx.y;
    const int xgid = blockIdx.x;
    const int ygid = blockIdx.y;

    // individual control variable in this work group and the work group size
    const int l = ylid * xlsz + xlid;
    const int lsz = xlsz * ylsz;

    // stride and maximum capacity of local buffer
    // cells of float4 per pixel with a surrounding border of 1 cell
    const int stride = xlsz + 2;
    const int maxbuf = stride * (ylsz + 2);

    // coordinates of top left pixel of buffer
    // this is 1 pixel left and above of the work group origin
    const int xul = xgid * xlsz - 1;
    const int yul = ygid * ylsz - 1;

    // populate local memory buffer
    for(int n = 0; n <= maxbuf/lsz; n++)
    {
        const int bufidx = n * lsz + l;
        if(bufidx >= maxbuf) continue;
        const int xx = xul + bufidx % stride;
        const int yy = yul + bufidx / stride;
        redblue_buffer[bufidx] = (xx >= 0 && yy >= 0 && xx < width && yy < height) ? input[yy * width + xx] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
    }

    // center buffer around current x,y-Pixel
    float4* centered_redblue = redblue_buffer + (ylid + 1) * stride + xlid + 1;

    __syncthreads();

    if(x >= width || y >= height) return;
    const int row = y;
    const int col = x;
    const int c = FC(row, col, filters);
    float4 color = centered_redblue[0];
    if(x == 0 || y == 0 || x == (width-1) || y == (height-1))
    {
        output[y * width + x] = make_float4(fmaxf(color.x, 0.0f), fmaxf(color.y, 0.0f), fmaxf(color.z, 0.0f), 1.0f);
        return;
    }

    if(c == 1 || c == 3)
    { // calculate red and blue for green pixels:
        // need 4-nbhood:
        const float4 nt = centered_redblue[-stride];
        const float4 nb = centered_redblue[ stride];
        const float4 nl = centered_redblue[-1];
        const float4 nr = centered_redblue[ 1];
        if(FC(row, col+1, filters) == 0) // red nb in same row
        {
            color.z = (nt.z + nb.z + 2.0f*color.y - nt.y - nb.y)*0.5f;
            color.x = (nl.x + nr.x + 2.0f*color.y - nl.y - nr.y)*0.5f;
        }
        else
        { // blue nb
            color.x = (nt.x + nb.x + 2.0f*color.y - nt.y - nb.y)*0.5f;
            color.z = (nl.z + nr.z + 2.0f*color.y - nl.y - nr.y)*0.5f;
        }
    }
    else
    {
        // get 4-star-nbhood:
        const float4 ntl = centered_redblue[-stride - 1];
        const float4 ntr = centered_redblue[-stride + 1];
        const float4 nbl = centered_redblue[ stride - 1];
        const float4 nbr = centered_redblue[ stride + 1];

        if(c == 0)
        { // red pixel, fill blue:
            const float diff1  = fabsf(ntl.z - nbr.z) + fabsf(ntl.y - color.y) + fabsf(nbr.y - color.y);
            const float guess1 = ntl.z + nbr.z + 2.0f*color.y - ntl.y - nbr.y;
            const float diff2  = fabsf(ntr.z - nbl.z) + fabsf(ntr.y - color.y) + fabsf(nbl.y - color.y);
            const float guess2 = ntr.z + nbl.z + 2.0f*color.y - ntr.y - nbl.y;
            if     (diff1 > diff2) color.z = guess2 * 0.5f;
            else if(diff1 < diff2) color.z = guess1 * 0.5f;
            else color.z = (guess1 + guess2)*0.25f;
        }
        else // c == 2, blue pixel, fill red:
        {
            const float diff1  = fabsf(ntl.x - nbr.x) + fabsf(ntl.y - color.y) + fabsf(nbr.y - color.y);
            const float guess1 = ntl.x + nbr.x + 2.0f*color.y - ntl.y - nbr.y;
            const float diff2  = fabsf(ntr.x - nbl.x) + fabsf(ntr.y - color.y) + fabsf(nbl.y - color.y);
            const float guess2 = ntr.x + nbl.x + 2.0f*color.y - ntr.y - nbl.y;
            if     (diff1 > diff2) color.x = guess2 * 0.5f;
            else if(diff1 < diff2) color.x = guess1 * 0.5f;
            else color.x = (guess1 + guess2)*0.25f;
        }
    }
    output[y * width + x] = make_float4(fmaxf(color.x, 0.0f), fmaxf(color.y, 0.0f), fmaxf(color.z, 0.0f), 1.0f);
}

/**
 * Demosaic image border - exact translation from darktable OpenCL
 */
__global__ void border_interpolate_kernel(
    float* input,
    float4* output,
    int width,
    int height,
    uint32_t filters,
    int border
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= width || y >= height) return;

    const int avgwindow = 1;

    if(x >= border && x < width-border && y >= border && y < height-border) return;

    float4 o;
    float sum[4] = { 0.0f };
    int count[4] = { 0 };

    for (int j=y-avgwindow; j<=y+avgwindow; j++) for (int i=x-avgwindow; i<=x+avgwindow; i++)
    {
        if (j>=0 && i>=0 && j<height && i<width)
        {
            const int f = FC(j,i,filters);
            sum[f] += fmaxf(0.0f, input[j * width + i]);
            count[f]++;
        }
    }

    const float i = fmaxf(0.0f, input[y * width + x]);
    o.x = count[0] > 0 ? sum[0]/count[0] : i;
    o.y = count[1]+count[3] > 0 ? (sum[1]+sum[3])/(count[1]+count[3]) : i;
    o.z = count[2] > 0 ? sum[2]/count[2] : i;

    const int f = FC(y,x,filters);

    if     (f == 0) o.x = i;
    else if(f == 1) o.y = i;
    else if(f == 2) o.z = i;
    else            o.y = i;

    output[y * width + x] = o;
}



// Complete demosaic function implementation - exact darktable flow
torch::Tensor ppg_demosaic_cuda(torch::Tensor input, uint32_t filters, float median_threshold) {
    // Input validation
    TORCH_CHECK(input.device().is_cuda(), "Input tensor must be on CUDA device");
    TORCH_CHECK(input.dtype() == torch::kFloat32, "Input tensor must be float32");
    TORCH_CHECK(input.dim() == 3, "Input tensor must be 3D (H, W, 1)");
    TORCH_CHECK(input.size(2) == 1, "Input must have single channel (raw Bayer)");
    
    // Ensure input is contiguous
    input = input.contiguous();
    
    const int height = input.size(0);
    const int width = input.size(1);
    
    // Create output tensor (H, W, 4) for RGBA
    auto output = torch::zeros({height, width, 4}, 
                              torch::TensorOptions().dtype(torch::kFloat32).device(input.device()));
    
    // Create temporary buffer for median filtering
    auto temp_median = torch::zeros({height, width}, 
                                   torch::TensorOptions().dtype(torch::kFloat32).device(input.device()));
    
    // Get CUDA stream
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    
    // Setup grid and block dimensions - must match darktable's local work group sizing
    dim3 block(16, 16);  // This needs to match OpenCL local size
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    
    // Calculate shared memory sizes for each kernel
    const int median_stride = block.x + 2*2;
    const int median_shared_size = median_stride * (block.y + 2*2) * sizeof(float);
    
    const int green_stride = block.x + 2*3;
    const int green_shared_size = green_stride * (block.y + 2*3) * sizeof(float);
    
    const int redblue_stride = block.x + 2;
    const int redblue_shared_size = redblue_stride * (block.y + 2) * sizeof(float4);
    
    auto* processing_input = input.data_ptr<float>();
    
    // Step 1: Border interpolation - demosaic border regions to RGB FIRST
    border_interpolate_kernel<<<grid, block, 0, stream>>>(
        input.data_ptr<float>(), reinterpret_cast<float4*>(output.data_ptr<float>()), 
        width, height, filters, 3);
    
    // Step 2: Pre-median filtering (optional) on raw data
    if (median_threshold > 0.0f) {
        pre_median_kernel<<<grid, block, median_shared_size, stream>>>(
            input.data_ptr<float>(), temp_median.data_ptr<float>(), 
            width, height, filters, median_threshold);
        processing_input = temp_median.data_ptr<float>();
    }
    
    // Step 3: Green channel interpolation - outputs to RGB buffer (overwrites borders)
    ppg_demosaic_green_kernel<<<grid, block, green_shared_size, stream>>>(
        processing_input, reinterpret_cast<float4*>(output.data_ptr<float>()), 
        width, height, filters);
    
    // Step 4: Red/Blue interpolation - final step using RGB buffer
    ppg_demosaic_redblue_kernel<<<grid, block, redblue_shared_size, stream>>>(
        reinterpret_cast<float4*>(output.data_ptr<float>()), 
        reinterpret_cast<float4*>(output.data_ptr<float>()), 
        width, height, filters);
    
    // Synchronize to ensure completion
    hipStreamSynchronize(stream);
    
    return output;
}
