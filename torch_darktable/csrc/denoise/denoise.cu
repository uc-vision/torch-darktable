#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#include "reduction.h"
#include "denoise.h"
#include "fft/fft_thread.h"
#include "window.h"
#include "cuda_utils.h"
#include "pixel.h"

#include <c10/cuda/CUDAStream.h>

namespace cg = cooperative_groups;

namespace {

constexpr float eps = 1e-15f;

// Fixed-size constant memory for noise sigmas (max 3 channels)
__constant__ float noise_sigmas[3];

// Helper to upload noise sigmas to constant memory
template<typename T>
inline void upload_noise_sigmas(const torch::Tensor& noise_sigmas_tensor, hipStream_t stream) {
    constexpr int C = channels<T>();
    CUDA_CHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(noise_sigmas), noise_sigmas_tensor.data_ptr<float>(), 
                                      C * sizeof(float), 0, hipMemcpyDeviceToDevice, stream));
}




// Using custom FFT implementation

__device__ __forceinline__ int2 get_group_pos() {
    auto block = cg::this_thread_block();
    dim3 g = block.group_index();
    return make_int2(g.x, g.y);
}

__device__ __forceinline__ int reflect_1d(int x, int limit) {
    if (x < 0) x = -x;
    if (x >= limit) x = 2 * limit - x - 1;
    return x;
}


template<int K, int C>
__device__ __forceinline__ void load_channel(
    const float* __restrict__ img,
    int stride, int H, int W,
    int col, int c, float chan_data[K]
) {
    int2 g = get_group_pos();
    int2 grid_offset = {K / stride, K / stride};
    
    int src_x = (g.x - grid_offset.x) * stride + col;
    int refl_x = reflect_1d(src_x, W);
    
    #pragma unroll
    for (int row = 0; row < K; row++) {
        int src_y = (g.y - grid_offset.y) * stride + row;
        int refl_y = reflect_1d(src_y, H);
        chan_data[row] = img[(refl_y * W + refl_x) * C + c];
    }
}

template<int K, int C>
__device__ __forceinline__ void store_channel(
    float* __restrict__ out,
    float* __restrict__ mask,
    int stride, int H_pad, int W_pad,
    int col, int c, const Complex chan_data[K],
    float chan_mean
) {
    int2 g = get_group_pos();
    int2 grid_offset = {K / stride, K / stride};
    
    int out_x = (g.x - grid_offset.x) * stride + col + K;    
    if (out_x >= W_pad) return;
    
    #pragma unroll
    for (int row = 0; row < K; row++) {
        int out_y = (g.y - grid_offset.y) * stride + row + K;
        
        if (out_y < H_pad) {
            int2 pos = make_int2(col, row);
            float fft_window = Window<K>::fft_window(pos);
            auto interp_window = Window<K>::interp_window(pos);
            
            int out_idx = out_y * W_pad + out_x;
            float value = chan_data[row].re;

            float reconstructed_chan = (value + chan_mean * fft_window) * interp_window;
            atomicAdd(&out[out_idx * C + c], reconstructed_chan);
            
            if (c == 0) {  // Only add mask once per pixel
                atomicAdd(&mask[out_idx], fft_window * interp_window);
            }
        }
    }
}


__device__ __forceinline__ Complex apply_gain(Complex value, float sigma) {
  float power = value.magnitude_squared() + eps;
  float gain = fmaxf(power - sigma * sigma, 0.0f) / power;
  return gain * value;
}



// Main kernel: orchestrates the Wiener filtering pipeline
template<int K, int C>
__global__ void wiener_tile_kernel(
    const float* __restrict__ img,
    float* __restrict__ out,
    float* __restrict__ mask,
    int H, int W, int H_pad, int W_pad,
    int stride,
    int channel
) {
    auto warp = cg::tiled_partition<32>(cg::this_thread_block());
    int col = warp.thread_rank();   // Thread processes column 'col' (0 to K-1)
    
    // Load single channel data
    float chan_data[K];
    load_channel<K, C>(img, stride, H, W, col, channel, chan_data);
    
    // Compute channel mean
    float chan_sum = 0.0f;
    #pragma unroll
    for (int row = 0; row < K; row++) {
        chan_sum += chan_data[row];
    }
    float tile_chan_sum = cg::reduce(warp, chan_sum, cg::plus<float>{});
    float chan_mean = tile_chan_sum / (K * K);
    
    // Apply windowing and subtract mean
    Complex fft_data[K];


    #pragma unroll
    for (int row = 0; row < K; row++) {
        int2 pos = make_int2(col, row);
        float fft_window = Window<K>::fft_window(pos);

        fft_data[row].re = (chan_data[row] - chan_mean) * fft_window;
        fft_data[row].im = 0.0f;
    }
        
    // FFT processing
    fft_2d<K>(fft_data);
    
    #pragma unroll
    for (int row = 0; row < K; row++) {
        fft_data[row] = apply_gain(fft_data[row], noise_sigmas[channel]);
    }
    
    ifft_2d<K>(fft_data);
    
    
    // Store channel back to output
    store_channel<K, C>(out, mask, stride, H_pad, W_pad, col, channel, fft_data, chan_mean);
}


template<int K, typename T>
__global__ void normalize_and_crop_kernel(
    const T* __restrict__ padded_out,  // (C, H_pad, W_pad)
    T* __restrict__ final_out,         // (C, H, W) - final output
    const float* __restrict__ mask,        // (H_pad, W_pad)

    int H, int W, int H_pad, int W_pad
) {
  
    int2 pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x, 
                         blockIdx.y * blockDim.y + threadIdx.y);
    
    if (pos.y < H && pos.x < W) {
        int2 pad_pos = pos + K;
        int padded_idx = pad_pos.y * W_pad + pad_pos.x;
        int idx = (pos.y * W + pos.x);  // HWC layout

        final_out[idx] = padded_out[padded_idx] / (mask[padded_idx] + eps);
    }
}


template<int K>
struct WienerImpl final : public Wiener {
    static_assert(K == 16 || K == 32, "K must be 16 or 32");

    torch::Device device_;
    int overlap_factor_;

public:
    WienerImpl(torch::Device device,
      int overlap_factor = 4,  
      const float interp_scale = 0.3f, const float fft_scale = 0.3f)
        : device_(device), overlap_factor_(overlap_factor) {
        
        auto stream = at::cuda::getCurrentCUDAStream(device.index());
        Window<K>::init(fft_scale, interp_scale, stream);
    }
    
private:
    template<int C>
    torch::Tensor _process(const torch::Tensor &input, const torch::Tensor &noise_sigmas) {
        static_assert(C == 1 || C == 3, "C must be 1 or 3");
        
        using pixel = std::conditional_t<C == 1, float, float3>;
        TORCH_CHECK(input.device() == device_, "input device mismatch");
        TORCH_CHECK(input.dim() == 3, "expected HWC tensor");

        const int H = static_cast<int>(input.size(0));
        const int W = static_cast<int>(input.size(1));
        const int input_C = static_cast<int>(input.size(2));
        TORCH_CHECK(input_C == C, "input channel count mismatch");
        
        // Compute dimensions on-demand
        const int stride = K / overlap_factor_;  // Configurable overlap
        const int h_pad = H + 2 * K;
        const int w_pad = W + 2 * K;
        
        // Grid needs to start earlier to cover boundaries with proper overlap
        const int grid_start = -(K / stride);  // Start early for boundary coverage
        const int grid_h = (H + K + stride - 1) / stride - grid_start;  // Extended coverage
        const int grid_w = (W + K + stride - 1) / stride - grid_start;
        
        auto padded_out = torch::zeros({h_pad, w_pad, C}, input.options());
        auto mask = torch::zeros({h_pad, w_pad}, input.options());
        auto final_out = torch::empty({H, W, C}, input.options());
        
        // Process noise sigmas
        TORCH_CHECK(noise_sigmas.numel() == C, "noise_sigmas must have C elements");
        
        dim3 grid(grid_w, grid_h);
        dim3 block(K);  // K threads, each handles one row of K pixels

        auto stream = at::cuda::getCurrentCUDAStream(device_.index());
        
        // Upload noise sigmas to constant memory
        upload_noise_sigmas<pixel>(noise_sigmas, stream);
       
        // Launch kernel once per channel
        for (int c = 0; c < C; c++) {
            wiener_tile_kernel<K, C><<<grid, block, 0, stream>>>(
                input.data_ptr<float>(),
                padded_out.data_ptr<float>(),
                mask.data_ptr<float>(),
                H, W, h_pad, w_pad,
                stride,
                c
            );
        }

        
        // Check for kernel launch errors
        auto cuda_err = hipGetLastError();
        TORCH_CHECK(cuda_err == hipSuccess, "Kernel launch failed: ", hipGetErrorString(cuda_err));
        
        // Normalize and crop in a single kernel
        dim3 norm_grid(div_up(W, K), div_up(H, K));
        dim3 norm_block(K, K);
        
        normalize_and_crop_kernel<K, pixel><<<norm_grid, norm_block, 0, stream>>>(
            reinterpret_cast<pixel const*>(padded_out.data_ptr<float>()),
            reinterpret_cast<pixel*>(final_out.data_ptr<float>()),
            mask.data_ptr<float>(),
            H, W, h_pad, w_pad
        );
        
        CUDA_CHECK_KERNEL();
        
        return final_out;
    }

public:
    torch::Tensor process(const torch::Tensor &input, const torch::Tensor &noise_sigmas) override {
        TORCH_CHECK(input.dim() == 3, "expected HWC tensor");
        const int input_C = static_cast<int>(input.size(2));
        
        if (input_C == 1) {
            return _process<1>(input, noise_sigmas);
        } else if (input_C == 3) {
            return _process<3>(input, noise_sigmas);
        } else {
            TORCH_CHECK(false, "input channels must be 1 or 3, got ", input_C);
        }
    }
    
    int get_overlap_factor() const override { return overlap_factor_; }
    

};

} // namespace

std::shared_ptr<Wiener> create_wiener(torch::Device device,
    int width, int height, int overlap_factor, int tile_size) {

    if (tile_size == 16) {
        return std::make_shared<WienerImpl<16>>(device,  overlap_factor);
    } else {
        return std::make_shared<WienerImpl<32>>(device,  overlap_factor);
    }
}





