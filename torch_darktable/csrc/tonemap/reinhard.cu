#include "hip/hip_runtime.h"
#include "../cuda_utils.h"
#include "../device_math.h"
#include "../reduction.h"

#include "color_adaption.h"
#include "tonemap.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <vector>

#include <c10/cuda/CUDAStream.h>

__device__ __constant__ ColorTransform transform;

// Reinhard tone mapping kernel
__global__ void reinhard_tonemap_kernel(
    const float* __restrict__ input,
    uint8_t* __restrict__ output,
    float gamma,
    int height,
    int width
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return;
       
    // Load input pixel
    int idx = y * width + x;
    float3 rgb = float3_load(input, idx);
    
    // Scale to [0,1] range
    float3 scaled = (rgb - transform.bounds_min) / transform.range;

    // Apply tone mapping
    float3 adapt = pow(transform.adapt_mean / transform.exposure, transform.map_key);
    float3 tonemapped = scaled / (adapt + scaled);    

    // Apply gamma correction and convert to 8-bit
    float3 gamma_corrected = pow(fmax(tonemapped, 0.0f), 1.0f / gamma);
    float3_to_uint8_rgb(gamma_corrected, output, idx);
}


torch::Tensor reinhard_tonemap(
    const torch::Tensor& image,
    const torch::Tensor& metrics,
    const TonemapParams& params
) {

    check_image(image);

    int height = image.size(0);
    int width = image.size(1);

    auto stream = at::cuda::getCurrentCUDAStream().stream();

    metrics_to_transform(transform, metrics, params, stream);
    
    auto output = torch::empty({height, width, 3}, torch::dtype(torch::kUInt8).device(image.device()));

    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                   (height + block_size.y - 1) / block_size.y);

    reinhard_tonemap_kernel<<<grid_size, block_size, 0, stream>>>(
        image.data_ptr<float>(),
        output.data_ptr<uint8_t>(),
        params.gamma,
        height,
        width
    );

    CUDA_CHECK_KERNEL();
    return output;
}
