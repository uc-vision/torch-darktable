#include "hip/hip_runtime.h"
/*
    This file is part of darktable,
    copyright (c) 2025 darktable developers.

    darktable is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    darktable is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with darktable.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <torch/torch.h>
#include <ATen/ATen.h>
#include <c10/cuda/CUDAStream.h>
#include <stdexcept>
#include <algorithm>

#include "../cuda_utils.h"
#include "../device_math.h"
#include "bilateral.h"



// Trilinear interpolation from 3D grid at base index gi with fractional coords f
__device__ __forceinline__ float trilerp(const float *g, int gi, float fx, float fy, float fz, int ox, int oy, int oz)
{
    const float ax = 1.0f - fx, ay = 1.0f - fy, az = 1.0f - fz;
    return g[gi]                 * ax * ay * az
         + g[gi + ox]            * fx * ay * az
         + g[gi + oy]            * ax * fy * az
         + g[gi + oy + ox]       * fx * fy * az
         + g[gi + oz]            * ax * ay * fz
         + g[gi + oz + ox]       * fx * ay * fz
         + g[gi + oz + oy]       * ax * fy * fz
         + g[gi + oz + oy + ox]  * fx * fy * fz;
}

__device__ __forceinline__ void trilerp_add(
    float *g, int gi, float fx, float fy, float fz, int ox, int oy, int oz, float scale)
{
    const float ax = 1.0f - fx, ay = 1.0f - fy, az = 1.0f - fz;
    const float bx = fx,        by = fy,        bz = fz;

    atomicAdd(g + gi,                ax * ay * az * scale);
    atomicAdd(g + gi + ox,           bx * ay * az * scale);
    atomicAdd(g + gi + oy,           ax * by * az * scale);
    atomicAdd(g + gi + oy + ox,      bx * by * az * scale);
    atomicAdd(g + gi + oz,           ax * ay * bz * scale);
    atomicAdd(g + gi + oz + ox,      bx * ay * bz * scale);
    atomicAdd(g + gi + oz + oy,      ax * by * bz * scale);
    atomicAdd(g + gi + oz + oy + ox, bx * by * bz * scale);
}

struct GridSample
{
    int gi, ox, oy, oz;
    float fx, fy, fz;
};

__device__ __forceinline__ GridSample make_grid_sample(
    int x, int y, float L,
    int sizex, int sizey, int sizez,
    float sigma_s, float sigma_r)
{
    const float3 g = make_float3(
        clampf(x / sigma_s, 0.0f, (float)(sizex - 1)),
        clampf(y / sigma_s, 0.0f, (float)(sizey - 1)),
        clampf(L / sigma_r, 0.0f, (float)(sizez - 1)));
    const int3 ib = min(make_int3(g), make_int3(sizex - 2, sizey - 2, sizez - 2));
    const float3 f = g - ib;
    GridSample s;
    s.gi = grid_index(ib.x, ib.y, ib.z, sizex, sizey, sizez);
    s.ox = 1;
    s.oy = sizex;
    s.oz = sizex * sizey;
    s.fx = f.x; s.fy = f.y; s.fz = f.z;
    return s;
}


// Zero out a 2D slice of the 3D grid interpreted as [sizex, sizey*sizez]
__global__ void zero_grid(float *grid, int width, int height)
{
    int2 pos = pixel_index();
    if(pos.x >= width || pos.y >= height) return;
    grid[pos.x + width * pos.y] = 0.0f;
}


// Splat kernel: builds the bilateral grid from luminance image
__global__ void splat_kernel(
    const float *in, float *grid,
    const int width, const int height,
    const int sizex, const int sizey, const int sizez,
    const float sigma_s, const float sigma_r)
{
    int2 pos = pixel_index();
    if(pos.x >= width || pos.y >= height) return;

    const float L = in[pos.y * width + pos.x];
    const GridSample s = make_grid_sample(pos.x, pos.y, L, sizex, sizey, sizez, sigma_s, sigma_r);
    const float contrib = 1.0f / (sigma_s * sigma_s);
    trilerp_add(grid, s.gi, s.fx, s.fy, s.fz, s.ox, s.oy, s.oz, contrib);
}

// Numerator splat: accumulate L * weights into grid
__global__ void splat_num_kernel(
    const float *in, float *grid,
    const int width, const int height,
    const int sizex, const int sizey, const int sizez,
    const float sigma_s, const float sigma_r)
{
    int2 pos = pixel_index();
    if(pos.x >= width || pos.y >= height) return;

    const float L = in[pos.y * width + pos.x];
    const GridSample s = make_grid_sample(pos.x, pos.y, L, sizex, sizey, sizez, sigma_s, sigma_r);
    const float contrib = 1.0f / (sigma_s * sigma_s);
    trilerp_add(grid, s.gi, s.fx, s.fy, s.fz, s.ox, s.oy, s.oz, contrib * L);
}


// Blur kernels, translated from bilateral.cl
__global__ void blur_line_kernel(
    const float *ibuf, float *obuf,
    const int offset1, const int offset2, const int offset3,
    const int size1, const int size2, const int size3)
{
    const int k = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(k >= size1 || j >= size2) return;

    const float w0 = 6.0f / 16.0f;
    const float w1 = 4.0f / 16.0f;
    const float w2 = 1.0f / 16.0f;

    int index = k * offset1 + j * offset2;

    float tmp1 = ibuf[index];
    obuf[index] = ibuf[index] * w0 + w1 * ibuf[index + offset3] 
                + w2 * ibuf[index + 2 * offset3];
    index += offset3;
    float tmp2 = ibuf[index];
    obuf[index] = ibuf[index] * w0 + w1 * (ibuf[index + offset3] + tmp1) 
                + w2 * ibuf[index + 2 * offset3];
    index += offset3;
    for(int i = 2; i < size3 - 2; i++)
    {
        const float tmp3 = ibuf[index];
        obuf[index] = ibuf[index] * w0 + w1 * (ibuf[index + offset3] + tmp2)
                    + w2 * (ibuf[index + 2 * offset3] + tmp1);
        index += offset3;
        tmp1 = tmp2;
        tmp2 = tmp3;
    }
    const float tmp3 = ibuf[index];
    obuf[index] = ibuf[index] * w0 + w1 * (ibuf[index + offset3] + tmp2) + w2 * tmp1;
    index += offset3;
    obuf[index] = ibuf[index] * w0 + w1 * tmp3 + w2 * tmp2;
}


__global__ void blur_line_z_kernel(
    const float *ibuf, float *obuf,
    const int offset1, const int offset2, const int offset3,
    const int size1, const int size2, const int size3)
{
    const int k = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(k >= size1 || j >= size2) return;

    const float w1 = 4.0f / 16.0f;
    const float w2 = 2.0f / 16.0f;

    int index = k * offset1 + j * offset2;

    float tmp1 = ibuf[index];
    obuf[index] = w1 * ibuf[index + offset3] + w2 * ibuf[index + 2 * offset3];
    index += offset3;
    float tmp2 = ibuf[index];
    obuf[index] = w1 * (ibuf[index + offset3] - tmp1) + w2 * ibuf[index + 2 * offset3];
    index += offset3;
    for(int i = 2; i < size3 - 2; i++)
    {
        const float tmp3 = ibuf[index];
        obuf[index] = + w1 * (ibuf[index + offset3]   - tmp2)
                      + w2 * (ibuf[index + 2 * offset3] - tmp1);
        index += offset3;
        tmp1 = tmp2;
        tmp2 = tmp3;
    }
    const float tmp3 = ibuf[index];
    obuf[index] = w1 * (ibuf[index + offset3] - tmp2) - w2 * tmp1;
    index += offset3;
    obuf[index] = - w1 * tmp3 - w2 * tmp2;
}


// Slice kernel: outputs processed luminance
__global__ void slice_kernel(
    const float *in, const float *grid, float *out,
    const int width, const int height,
    const int sizex, const int sizey, const int sizez,
    const float sigma_s, const float sigma_r,
    const float detail)
{
    int2 pos = pixel_index();
    if(pos.x >= width || pos.y >= height) return;

    const float L = in[pos.y * width + pos.x];

    // Scale matches removal of 100x in splat: 100 * 0.04 = 4
    const float norm = -detail * sigma_r * 4.0f;

    const GridSample s = make_grid_sample(pos.x, pos.y, L, sizex, sizey, sizez, sigma_s, sigma_r);
    const float Ldiff = trilerp(grid, s.gi, s.fx, s.fy, s.fz, s.ox, s.oy, s.oz);

    const float Lout = fmaxf(0.0f, L + norm * Ldiff);
    out[pos.y * width + pos.x] = Lout;
}

// Denoise: slice weighted average from two blurred grids (sum_wL, sum_w)
__global__ void slice_denoise_kernel(
    const float *in, const float *grid_num, const float *grid_den, float *out,
    const int width, const int height,
    const int sizex, const int sizey, const int sizez,
    const float sigma_s, const float sigma_r, const float amount)
{
    int2 pos = pixel_index();
    if(pos.x >= width || pos.y >= height) return;

    const float L = in[pos.y * width + pos.x];
    const GridSample s = make_grid_sample(pos.x, pos.y, L, sizex, sizey, sizez, sigma_s, sigma_r);
    const float num = trilerp(grid_num, s.gi, s.fx, s.fy, s.fz, s.ox, s.oy, s.oz);
    const float den = trilerp(grid_den, s.gi, s.fx, s.fy, s.fz, s.ox, s.oy, s.oz);

    const float denoised = (den > 1e-8f) ? (num / den) : L;
    const float Lout = (1.0f - amount) * L + amount * denoised;
    out[pos.y * width + pos.x] = Lout;
}


// Bilateral workspace implementation (style similar to Laplacian)
struct BilateralImpl : public Bilateral
{
    torch::Device device_;
    int width, height;
    float sigma_s, sigma_r;

    // Cache tensors for efficiency; infer grid dims from current parameters when (re)allocating
    torch::Tensor dev_grid;
    torch::Tensor dev_grid_tmp;
    torch::Tensor dev_grid_num; // reserved
    torch::Tensor dev_grid_den; // reserved

    BilateralImpl(torch::Device device,
                  int width, int height,
                  float sigma_s, float sigma_r)
        : device_(device), width(width), height(height),
          sigma_s(sigma_s), sigma_r(sigma_r)
    {
        TORCH_CHECK(width > 0 && height > 0, "Invalid dimensions");
    }

    std::tuple<int,int,int> compute_grid_size() const
    {
        float ss = sigma_s;
        if(ss < 0.5f) ss = 0.5f;

        // L range assumes input luminance in [0,1]
        const float L_range = 1.0f;

        float gx = roundf(width  / ss);
        float gy = roundf(height / ss);
        float gz = roundf(L_range / sigma_r);
        gx = fminf(fmaxf(gx, 4.0f), 3000.0f);
        gy = fminf(fmaxf(gy, 4.0f), 3000.0f);
        gz = fminf(fmaxf(gz, 4.0f), 50.0f);

        // Effective sigmas after potential clamping
        const float eff_sigma_s = fmaxf(height / gy, width / gx);
        const float eff_sigma_r = L_range / gz;
        const float s_s = eff_sigma_s;
        const float s_r = eff_sigma_r;
        const int sx = (int)ceilf(width  / s_s) + 1;
        const int sy = (int)ceilf(height / s_s) + 1;
        const int sz = (int)ceilf(L_range / s_r) + 1;
        return {sx, sy, sz};
    }

    void invalidate_buffers()
    {
        dev_grid = torch::Tensor();
        dev_grid_tmp = torch::Tensor();
        dev_grid_num = torch::Tensor();
        dev_grid_den = torch::Tensor();
    }

    void check_input_image(const torch::Tensor &luminance) const
    {
        TORCH_CHECK(luminance.dtype() == torch::kFloat32, "Input must be float32");
        TORCH_CHECK(luminance.dim() == 2, "Input must be 2D (H,W)");
        TORCH_CHECK(luminance.size(0) == height && luminance.size(1) == width,
                    "Input dims must match (H,W)");
        TORCH_CHECK(luminance.is_cuda(), "Input must be CUDA tensor");
    }

    // Host helpers for blur scheduling
    void blur_xy(torch::Tensor &src, torch::Tensor &tmp, torch::Tensor &y_out, 
                 int sx, int sy, int sz, hipStream_t stream)
    {
        blur_line_kernel<<<grid2d(sz, sy), block_size_2d, 0, stream>>>(
            src.data_ptr<float>(), tmp.data_ptr<float>(), sx * sy, sx, 1, sz, sy, sx);
        blur_line_kernel<<<grid2d(sz, sx), block_size_2d, 0, stream>>>(
            tmp.data_ptr<float>(), y_out.data_ptr<float>(), sx * sy, 1, sx, sz, sx, sy);
    }

    void blur_z_gaussian(torch::Tensor &y_in, torch::Tensor &z_out, 
                         int sx, int sy, int sz, hipStream_t stream)
    {
        blur_line_kernel<<<grid2d(sx, sy), block_size_2d, 0, stream>>>(
            y_in.data_ptr<float>(), z_out.data_ptr<float>(), 1, sx, sx * sy, sx, sy, sz);
    }

    void blur_z_derivative(torch::Tensor &y_in, torch::Tensor &z_out, 
                           int sx, int sy, int sz, hipStream_t stream)
    {
        blur_line_z_kernel<<<grid2d(sx, sy), block_size_2d, 0, stream>>>(
            y_in.data_ptr<float>(), z_out.data_ptr<float>(), 1, sx, sx * sy, sx, sy, sz);
    }

    // Ensure buffers for detail mode (single grid + tmp)
    void ensure_detail_buffers(int sx, int sy, int sz)
    {
        auto opts = torch::TensorOptions().device(device_).dtype(torch::kFloat32);
        if (!dev_grid.defined()) {
            dev_grid = torch::empty({sz, sy, sx}, opts);
            dev_grid_tmp = torch::empty({sz, sy, sx}, opts);
        }
    }

    // removed denoise buffers

    

    torch::Tensor process(const torch::Tensor &luminance, float detail) override
    {
        check_input_image(luminance);

        auto stream = c10::cuda::getCurrentCUDAStream();

        // Compute grid sizes on-the-fly and (re)allocate cached buffers if needed
        auto [sx, sy, sz] = compute_grid_size();
        ensure_detail_buffers(sx, sy, sz);
        dev_grid.zero_();

  
        splat_kernel<<<grid2d(width, height), block_size_2d, 0, stream.stream()>>>(
            luminance.data_ptr<float>(), dev_grid.data_ptr<float>(),
            width, height, sx, sy, sz, sigma_s, sigma_r);
  
        // Blur passes via helpers
        blur_xy(dev_grid, dev_grid_tmp, dev_grid, sx, sy, sz, stream.stream());
        blur_z_derivative(dev_grid, dev_grid_tmp, sx, sy, sz, stream.stream());

        // Slice
        auto output = torch::empty({height, width}, luminance.options());
        slice_kernel<<<grid2d(width, height), block_size_2d, 0, stream.stream()>>>(
            luminance.data_ptr<float>(), dev_grid_tmp.data_ptr<float>(), output.data_ptr<float>(),
            width, height, sx, sy, sz, sigma_s, sigma_r, detail);

        return output;
    }

    

    void set_sigma_s(float v) override { sigma_s = v; invalidate_buffers(); }
    void set_sigma_r(float v) override { sigma_r = v; invalidate_buffers(); }
    float get_sigma_s() const override { return sigma_s; }
    float get_sigma_r() const override { return sigma_r; }
};


std::shared_ptr<Bilateral> create_bilateral(
    torch::Device device,
    int width, int height,
    float sigma_s, float sigma_r) {
    return std::make_shared<BilateralImpl>(
      device, width, height, sigma_s, sigma_r);
}


